#include "hip/hip_runtime.h"
/**
 * Copyright (C) 2008-2012 Zillians, Inc. <http://www.zillians.com/>
 *
 * This program is free software: you can redistribute it and/or modify it
 * under the terms of the GNU Affero General Public License, version 3,
 * as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
 * or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Affero General Public License for more details.
 *
 * You should have received a copy of the GNU Affero General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "thor/lang/Language.cuh"
#include <stdio.h>

namespace thor {

namespace lang {

namespace internal {

//#define CEILING(x,y) (((x) + (y) - 1) / (y))

//#define MAX_OBJECT_COUNT 32*32*32*32  // 1M objects
//#define TOTAL_FLAGS  (1+32+32*32+32*32*32)

const int64_t cMagicNumber = 0xFFEEFFAABBEE9933;

struct TrackerHeader
{
	uint64 epoch;
    uint64 type_id;
    uint64 session_id;
    uint64 database_id;
    uint32 allocation_level; // TODO change to size
	uint32 allocation_index;
	char* ptr;
	bool is_valid;
};

struct ObjectHeader
{
    int64 magic_number;
    TrackerHeader* tracker_ptr;
};

struct Block64 {
	uint64 t0;
	uint64 t1;
	uint64 t2;
	uint64 t3;
	uint64 t4;
	uint64 t5;
	uint64 t6;
	uint64 t7;
};

struct Block128 {
	uint64 t0;
	uint64 t1;
	uint64 t2;
	uint64 t3;
	uint64 t4;
	uint64 t5;
	uint64 t6;
	uint64 t7;
	uint64 t8;
	uint64 t9;
	uint64 t10;
	uint64 t11;
	uint64 t12;
	uint64 t13;
	uint64 t14;
	uint64 t15;
};

__device__ TrackerHeader* header_blocks;
__device__ uint32* header_bit_flags; // size = 1 + 32 + 32*32 + 32*32*32

__global__ void __configureHeaderBlocksKernel(TrackerHeader* blocks, uint32* bit_flags)
{
	header_blocks = blocks;
	header_bit_flags = bit_flags;
}

void __configureHeaderBlocks(TrackerHeader* blocks, uint32* bit_flags)
{
	__configureHeaderBlocksKernel<<<1,1>>>(blocks, bit_flags);
}

__device__ Block64* b64_blocks;
__device__ uint32* b64_bit_flags; // size = 1 + 32 + 32*32 + 32*32*32

__global__ void __configureBlocksB64Kernel(Block64* blocks, uint32* bit_flags)
{
	b64_blocks = blocks;
	b64_bit_flags = bit_flags;
}

void __configureBlocksB64(Block64* blocks, uint32* bit_flags)
{
	__configureBlocksB64Kernel<<<1,1>>>(blocks, bit_flags);
}

__device__ Block128* b128_blocks;
__device__ uint32* b128_bit_flags; // size = 1 + 32 + 32*32 + 32*32*32

__global__ void __configureBlocksB128Kernel(Block128* blocks, uint32* bit_flags)
{
	b128_blocks = blocks;
	b128_bit_flags = bit_flags;
}

void __configureBlocksB128(Block128* blocks, uint32* bit_flags)
{
	__configureBlocksB128Kernel<<<1,1>>>(blocks, bit_flags);
}

#define ATOMIC_CASCADE_FLAGS 1

template<typename T>
__device__ T* __allocateBlock(uint32* bit_flags, T* blocks)
{
	// TODO rewrite in PTX assembly to further improve the performance
	uint32 tid = blockDim.x * blockIdx.x + threadIdx.x;

	uint32 f0 = bit_flags[0];
	uint32 f0t;
	uint32 msb0;

	f0t = (tid & 0x1U) ? __brev(f0) : f0;

	while(true)
	{
		asm("bfind.u32 %0, %1;\n\t" : "=r"(msb0) : "r"(f0t));

		if(msb0 == 0xFFFFFFFF)
			break;
		else
		{
			if(tid & 0x1U) msb0 = 31 - msb0;

			uint32 offset1 = 1 + msb0;
			uint32 f1 = bit_flags[offset1];
			uint32 f1t;
			uint32 msb1;

			f1t = (tid & 0x2U) ? __brev(f1) : f1;

			while(true)
			{
				asm("bfind.u32 %0, %1;\n\t" : "=r"(msb1) : "r"(f1t));

				if(msb1 == 0xFFFFFFFF)
					break;
				else
				{
					if(tid & 0x2U) msb1 = 31 - msb1;

					uint32 offset2 = 1 + 32 + msb0 * 32 + msb1;
					uint32 f2 = bit_flags[offset2];
					uint32 f2t;
					uint32 msb2;

					f2t = (tid & 0x4U) ? __brev(f2) : f2;

					while(true)
					{
						asm("bfind.u32 %0, %1;\n\t" : "=r"(msb2) : "r"(f2t));

						if(msb2 == 0xFFFFFFFF)
							break;
						else
						{
							if(tid & 0x4U) msb2 = 31 - msb2;

							uint32 offset3 = 1 + 32 + 32 * 32 + msb0 * 32 * 32 + msb1 * 32 + msb2;
							uint32 f3 = bit_flags[offset3];
							uint32 f3t;
							uint32 msb3;

							f3t = (tid & 0x8U) ? __brev(f3) : f3;

							while(true)
							{
								asm("bfind.u32 %0, %1;\n\t" : "=r"(msb3) : "r"(f3t));

								if(msb3 == 0xFFFFFFFF)
									break;
								else
								{
									if(tid & 0x8U) msb3 = 31 - msb3;

									uint32 index = 32 * 32 * 32 * msb0 + 32 * 32 * msb1 + 32 * msb2 + msb3;
									uint32 mask = 0xFFFFFFFF ^ (0x1U << (msb3));
									//printf("[%d] index = %d, mask = %x\n", tid, index, mask);

									uint32 old_flag = atomicAnd(&bit_flags[offset3], mask);
									uint32 result = old_flag & (~mask);
									if(result != 0x0U)
									{
										//printf("%d -> %d -> (%02d, %02d, %02d, %02d)\n", tid, index, msb0, msb1, msb2, msb3);
										//printf("%d: allocated msb0 = %u, offset1 = %u, f1t = %u, msb1 = %u, offset2 = %u, f2t = %u, msb2 = %u, offset3 = %u, f3t = %u, msb3 = %u, mask = %x\n", tid, msb0, offset1, f1t, msb1, offset2, f2t, msb2, offset3, f3t, msb3, mask);
										return &blocks[index];
									}
									else
									{
										//printf("%d: atomicAnd failed, old_flag = %x, result = %x\n", tid, old_flag, result);
										//f3 = bit_flags[offset3];
										f3 = old_flag;
										f3t = (tid & 0x8U) ? __brev(f3) : f3;
									}
								}
								//f3 ^= 0x1U << (msb3);
							}
#if !ATOMIC_CASCADE_FLAGS
							if(tid & 0x4U) msb2 = 31 - msb2;
#endif
						}

#if ATOMIC_CASCADE_FLAGS
						f2 ^= 0x1U << (msb2);
						uint32 t = atomicAnd(&bit_flags[offset2], f2);
						f2 = t & f2;
						f2t = (tid & 0x4U) ? __brev(f2) : f2;
#else
						f2t ^= 0x1U << (msb2);
#endif
					}
#if !ATOMIC_CASCADE_FLAGS
					if(tid & 0x2U) msb1 = 31 - msb1;
#endif
				}

#if ATOMIC_CASCADE_FLAGS
				f1 ^= 0x1U << (msb1);
				uint32 t = atomicAnd(&bit_flags[offset1], f1);
				f1 = t & f1;
				f1t = (tid & 0x2U) ? __brev(f1) : f1;
#else
				f1t ^= 0x1U << (msb1);
#endif
			}

#if !ATOMIC_CASCADE_FLAGS
			if(tid & 0x1U) msb0 = 31 - msb0;
#endif
		}

#if ATOMIC_CASCADE_FLAGS
		f0 ^= 0x1U << (msb0);
		uint32 t = atomicAnd(&bit_flags[0], f0);
		f0 = t & f0;
		f0t = (tid & 0x1U) ? __brev(f0) : f0;
#else
		f0t ^= 0x1U << (msb0);
#endif
	}

	return NULL;
}

template<typename T>
__device__ void __deallocateBlock(uint32* bit_flags, T* blocks, T* block)
{
	//uint32 tid = blockDim.x * blockIdx.x + threadIdx.x;

	uint32 index = (uint32)(block - &blocks[0]);
#if ATOMIC_CASCADE_FLAGS
	uint32 local_offset3 = (index / 32);
	uint32 local_offset2 = (local_offset3 / 32);
	uint32 local_offset1 = (local_offset2 / 32);
	//uint32 local_offset0 = (local_offset1 / 32);

	uint32 msb3 = index % 32;
	uint32 msb2 = local_offset3 % 32;
	uint32 msb1 = local_offset2 % 32;
	uint32 msb0 = local_offset1 % 32;
	uint32 local_mask3 = 0x1U << msb3;
	uint32 local_mask2 = 0x1U << msb2;
	uint32 local_mask1 = 0x1U << msb1;
	uint32 local_mask0 = 0x1U << msb0;

	uint32 offset0 = 0;
	uint32 offset1 = 1 + local_offset1;
	uint32 offset2 = 1 + 32 + local_offset2;
	uint32 offset3 = 1 + 32 + 32 * 32 + local_offset3;
	//printf("%d: deallocated msb0 = %u, offset1 = %u, msb1 = %u, offset2 = %u, msb2 = %u, offset3 = %u, msb3 = %u, mask = %x\n", tid, msb0, offset1, msb1, offset2, msb2, offset3, msb3, local_mask3);
	atomicOr(&bit_flags[offset0], local_mask0);
	atomicOr(&bit_flags[offset1], local_mask1);
	atomicOr(&bit_flags[offset2], local_mask2);
	atomicOr(&bit_flags[offset3], local_mask3);
#else
	uint32_t bucket = index / 32;
	uint32_t mask = 0x1 << (index % 32);
	uint32_t offset3 = 1 + 32 + 32 * 32 + bucket;
	atomicOr(&bit_flags[offset3], mask);
#endif
}

__device__ char* global_heap;
__device__ uint32 global_heap_size;
__device__ uint32 global_heap_bucket_size;
__device__ uint32* global_heap_bucket;
__device__ uint32* global_heap_bucket_offset;
__device__ unsigned long long int global_heap_next_offset;

__global__ void __configureGlobalHeapKernel(char* heap, uint32 heap_size, uint32* heap_bucket, uint32 heap_bucket_size, uint32* heap_bucket_offset, unsigned long long int heap_next_offset)
{
	global_heap = heap;
	global_heap_size = heap_size;
	global_heap_bucket_size = heap_bucket_size;
	global_heap_bucket = heap_bucket;
	global_heap_bucket_offset = heap_bucket_offset;
	global_heap_next_offset = heap_next_offset;
}

// exported and will be called from object service
void __configureGlobalHeap(char* heap, uint32 heap_size, uint32* heap_bucket, uint32 heap_bucket_size, uint32* heap_bucket_offset, unsigned long long int heap_next_offset)
{
	__configureGlobalHeapKernel<<<1,1>>>(
			heap,
			heap_size,
			heap_bucket,
			heap_bucket_size,
			heap_bucket_offset,
			heap_next_offset);
}

__device__ char* __allocateFromGlobalHeap(uint32 size, TrackerHeader* header)
{
	//uint32 tid = blockDim.x * blockIdx.x + threadIdx.x;

	//size += sizeof(uint64);
	size = (size+127) & (~0x7f); // round-up to next multiple of 128 to avoid false cache line sharing
	//printf("%d -> allocate %d bytes\n", tid, size);
	unsigned long long int offset_mask = (0x1ULL << 32) | size;
//	printf("%d -> mask = %llu\n", tid, offset_mask);
	unsigned long long int offset = atomicAdd(&global_heap_next_offset, offset_mask);

	// TODO check overflow
	uint32 last_allocated_size = (uint32)((offset << 32) >> 32);
	uint32 last_bucket_index = (uint32)(offset >> 32);

//	printf("%d -> last_allocated_size = %u, last_bucket_index = %u\n", tid, last_allocated_size, last_bucket_index);
	if((unsigned long long int)last_allocated_size + (unsigned long long int)size > (unsigned long long int)global_heap_size || last_bucket_index >= global_heap_bucket_size)
	{
		// TODO ?
		//printf("ERROR!!!\n");
		//atomicSub(&global_heap_next_offset, offset_mask);
		return NULL;
	}
	else
	{
		char* allocated_ptr = global_heap + last_allocated_size;
		global_heap_bucket[last_bucket_index] = size / 128;
		global_heap_bucket_offset[last_bucket_index] = last_allocated_size / 128;
		header->allocation_index = last_bucket_index;
		return allocated_ptr;
	}
}

__device__ void __deallocateFromGlobalHeap(char* ptr)
{
	global_heap_bucket[((ObjectHeader*)ptr)->tracker_ptr->allocation_index] = 0;
}

}

__device__ Object* __createObject(int64 size, int64 type_id)
{
	using namespace internal;

	//printf("header_bit_flags = %p\nheader_blocks = %p\n", header_bit_flags, header_blocks);
	TrackerHeader* header = __allocateBlock<TrackerHeader>(header_bit_flags, header_blocks);
	if(!header) return NULL;

	header->type_id = type_id;

	size += sizeof(ObjectHeader);

	char* memory_block = NULL;
	if(size <= 64)
	{
		header->allocation_level = 0;
		memory_block = (char*)__allocateBlock<Block64>(b64_bit_flags, b64_blocks);
	}
	else if(size <= 128)
	{
		header->allocation_level = 1;
		memory_block = (char*)__allocateBlock<Block128>(b128_bit_flags, b128_blocks);
	}
	else
	{
		header->allocation_level = 2;
		memory_block = __allocateFromGlobalHeap(size, header);
	}

	((ObjectHeader*)memory_block)->magic_number = cMagicNumber;
	((ObjectHeader*)memory_block)->tracker_ptr = header;

	header->ptr = memory_block;
	Object* result = (Object*)(memory_block + sizeof(ObjectHeader));

	//printf("creating object, size = %ld, type = %ld, ptr = %p\n", size, type_id, result);

	return result;
}


__device__ void __destroyObject(Object* object)
{
	using namespace internal;

	char* ptr = (char*)object;
	char* memory_block = ptr - sizeof(ObjectHeader);
	TrackerHeader* header = ((ObjectHeader*)memory_block)->tracker_ptr;

	// deallocate the physical memory
	if(header->allocation_level == 0)
	{
		__deallocateBlock<Block64>(b64_bit_flags, b64_blocks, (Block64*)memory_block);
	}
	else if(header->allocation_level == 1)
	{
		__deallocateBlock<Block128>(b128_bit_flags, b128_blocks, (Block128*)memory_block);
	}
	else
	{
		__deallocateFromGlobalHeap(memory_block);
	}

	// deallocate the tracker header
	__deallocateBlock<TrackerHeader>(header_bit_flags, header_blocks, header);
}


__device__ Object::Object() : mInternalDomain(NULL)
{

}

__device__ Object::~Object()
{ }

__device__ Object* Object::clone()
{
	return NULL;
}

__device__ int64 Object::hash()
{
	return 0LL;
}

__device__ Domain* Object::domain()
{
	return NULL;
}

} }
