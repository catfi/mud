#include "hip/hip_runtime.h"
/**
 * Copyright (C) 2008-2012 Zillians, Inc. <http://www.zillians.com/>
 *
 * This program is free software: you can redistribute it and/or modify it
 * under the terms of the GNU Affero General Public License, version 3,
 * as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful, but
 * WITHOUT ANY WARRANTY; without even the implied warranty of MERCHANTABILITY
 * or FITNESS FOR A PARTICULAR PURPOSE.
 *
 * See the GNU Affero General Public License for more details.
 *
 * You should have received a copy of the GNU Affero General Public License
 * along with this program. If not, see <http://www.gnu.org/licenses/>.
 */

#include "thor/lang/Language.cuh"
#include "thor/lang/Process.cuh"
#include "thor/lang/Domain.cuh"
//#include "thor/lang/Domain.h"
//#include "thor/lang/Lambda.h"
//#include "thor/container/Vector.h"
#include "framework/service/runtime/RuntimeServiceBuffer.h"
#include "framework/service/runtime/RuntimeServiceBufferCUDA.h"

using namespace thor;

namespace thor {

namespace lang {

namespace internal {

int32 gh_current_invocation_buffer_size = 0;
__device__ int32 gd_current_invocation_buffer_size;
zillians::framework::service::Invocation* gh_current_invocation_buffer_ptr = NULL;
__device__ zillians::framework::service::Invocation* gd_current_invocation_buffer_ptr;

__device__ int32* gd_next_invocation_buffer_index;
__device__ int32* gd_next_invocation_buffer_size;
__device__ zillians::framework::service::Invocation** gd_next_invocation_buffer_ptr;

int32 gh_current_invocation_shuffle_indices_size = 0;
int32* gh_current_invocation_shuffle_indices_ptr = NULL;
__device__ int32* gd_current_invocation_shuffle_indices_ptr;

#define __INVOCATION_PARAMETER_ELEMENTS   (CUDA_INVOCATION_PARAMETER_SIZE/CUDA_INVOCATION_PARAMETER_LOAD_SIZE)
#define __TOTAL_PARAMETER_SIZE_PER_BLOCK  (CUDA_DISPATCHER_THREADS_PER_BLOCK*CUDA_INVOCATION_PARAMETER_SIZE)
#define __TOTAL_LOAD_SIZE_PER_WARP        (CUDA_DISPATCHER_WARP_SIZE * CUDA_INVOCATION_PARAMETER_LOAD_SIZE)
#define __TOTAL_LOADER_WARP_COUNT         (__TOTAL_PARAMETER_SIZE_PER_BLOCK/__TOTAL_LOAD_SIZE_PER_WARP)
#define __TOTAL_WARPS_PER_INVOCATION      (CUDA_INVOCATION_PARAMETER_SIZE/__TOTAL_LOAD_SIZE_PER_WARP)
#define __TOTAL_WARPS_PER_BLOCK           (CUDA_DISPATCHER_THREADS_PER_BLOCK/CUDA_DISPATCHER_WARP_SIZE)
#define __TOTAL_WORKING_ITEMS_PER_WARP    CEILING(__TOTAL_LOADER_WARP_COUNT,__TOTAL_WARPS_PER_BLOCK)


#define CUDA_DMA_EMPTY_BARRIER 1
#define CUDA_DMA_FULL_BARRIER  2

//typedef char int8;
//typedef short int16;
//typedef int int32;
//typedef long long int64;

template<int Size>
struct type_selector;

template<>
struct type_selector<1> {
	typedef int8 type;
};

template<>
struct type_selector<2> {
	typedef int16 type;
};

template<>
struct type_selector<4> {
	typedef int32 type;
};

template<>
struct type_selector<8> {
	typedef int64 type;
};

typedef type_selector<CUDA_INVOCATION_PARAMETER_LOAD_SIZE>::type param_element_type;

__shared__ char __parameter_shm[CUDA_INVOCATION_PARAMETER_SIZE*CUDA_COMPUTE_THREADS_PER_BLOCK];

typedef type_selector<CUDA_INVOCATION_DMA_TRANSFER_SIZE>::type dma_element_type;

__shared__ int32 __dma_size[CUDA_COMPUTE_THREADS_PER_BLOCK];
__shared__ dma_element_type __dma_buffer[CUDA_INVOCATION_DMA_BUFFER_SIZE_PER_THREAD*CUDA_COMPUTE_THREADS_PER_BLOCK/CUDA_INVOCATION_DMA_TRANSFER_SIZE];
__shared__ dma_element_type* __dma_dest[CUDA_COMPUTE_THREADS_PER_BLOCK];
__shared__ dma_element_type* __dma_source[CUDA_COMPUTE_THREADS_PER_BLOCK];

//__device__ int32* __dma_size;
//__device__  dma_element_type* __dma_buffer;
//__device__  dma_element_type** __dma_dest;
//__device__  dma_element_type** __dma_source;

__global__ void __setCurrentInvocationBufferKernel(zillians::framework::service::Invocation* ptr, int32 size) {
	gd_current_invocation_buffer_ptr = ptr;
	gd_current_invocation_buffer_size = size;
}

// exported and will be called from runtime service
void __setCurrentInvocationBuffer(zillians::framework::service::Invocation* ptr, int32 size)
{
	if(gh_current_invocation_buffer_ptr != ptr || gh_current_invocation_buffer_size != size)
	{
		__setCurrentInvocationBufferKernel<<<1,1>>>(ptr, size);
		gh_current_invocation_buffer_ptr = ptr;
		gh_current_invocation_buffer_size = size;
	}
}

__global__ void __setNextInvocationBufferKernel(int32 target_id, int32 bucket_id, zillians::framework::service::Invocation* ptr, int32 size) {
	int32 index = target_id * CUDA_INVOCATION_NEXT_BUCKET_SIZE + bucket_id;
//	printf("setting next invocation buffer (%d,%d) to %p, size = %d\n", target_id, bucket_id, ptr, size);
//	printf("gd_next_invocation_buffer_ptr = %p\n", gd_next_invocation_buffer_ptr);
//	printf("gd_next_invocation_buffer_index = %p\n", gd_next_invocation_buffer_index);
//	printf("gd_next_invocation_buffer_size = %p\n", gd_next_invocation_buffer_size);
	gd_next_invocation_buffer_ptr[index] = ptr;
	gd_next_invocation_buffer_index[index] = 0;
	gd_next_invocation_buffer_size[index] = size;
}

void __setNextInvocationBuffer(int32 target_id, int32 bucket_id, zillians::framework::service::Invocation* ptr, int32 size) {
	__setNextInvocationBufferKernel<<<1,1>>>(target_id, bucket_id, ptr, size);
}

__global__ void __setNextInvocationBufferContainerKernel(zillians::framework::service::Invocation** ptr) {
//	printf("setting invocation buffer container to %p\n", ptr);
	gd_next_invocation_buffer_ptr = ptr;
}

void __setNextInvocationBufferContainer(zillians::framework::service::Invocation** ptr) {
	__setNextInvocationBufferContainerKernel<<<1,1>>>(ptr);
}

__global__ void __setNextInvocationSizeBufferKernel(int32* ptr) {
//	printf("setting invocation size buffer to %p\n", ptr);
	gd_next_invocation_buffer_size = ptr;
}

void __setNextInvocationSizeBuffer(int32* ptr) {
	__setNextInvocationSizeBufferKernel<<<1,1>>>(ptr);
}

__global__ void __setNextInvocationIndexBufferKernel(int32* ptr) {
//	printf("setting invocation index buffer to %p\n", ptr);
	gd_next_invocation_buffer_index = ptr;
}

void __setNextInvocationIndexBuffer(int32* ptr) {
	__setNextInvocationIndexBufferKernel<<<1,1>>>(ptr);
}

__global__ void __setShuffleIndicesBufferKernel(int32* ptr) {
	gd_current_invocation_shuffle_indices_ptr = ptr;
}

// exported and will be called from runtime service
void __setShuffleIndicesBuffer(int32* ptr, int32 size)
{
	if(gh_current_invocation_shuffle_indices_ptr != ptr)
	{
		__setShuffleIndicesBufferKernel<<<1,1>>>(ptr);
		gh_current_invocation_shuffle_indices_ptr = ptr;
	}
	gh_current_invocation_shuffle_indices_size = size;
}

__device__ __forceinline__ void __barrierSync(const int name, const int num)
{
	asm volatile("bar.sync %0, %1;" : : "r"(name), "r"(num) : "memory" );
}

__device__ __forceinline__ void __barrierArrive(const int name, const int num)
{
	asm volatile("bar.arrive %0, %1;" : : "r"(name), "r"(num) : "memory" );
}

__device__ bool __debugging_cond;
__global__ void __setDebugCondKernel(bool cond) {
	__debugging_cond = cond;
}

void __setDebugCond(bool cond) {
	__setDebugCondKernel<<<1,1>>>(cond);
}

__device__ zillians::framework::service::ControlFlags* __control_flags;
__global__ void __setControlFlagsKernel(zillians::framework::service::ControlFlags* flags) {
    __control_flags = flags;

    // initialize the flags
    __control_flags->daemonized = false;
    __control_flags->exited = false;
    __control_flags->exit_code = 0;
    __control_flags->explicit_exited = false;
    __control_flags->explicit_exit_code = 0;
    __control_flags->implicit_exited = false;
    __control_flags->implicit_exit_code = 0;
}

void __setControlFlags(zillians::framework::service::ControlFlags* flags) {
    __setControlFlagsKernel<<<1,1>>>(flags);
}

//__device__ void __loadParametersToShm(char* __parameter_shm, int32* __dma_size, dma_element_type* __dma_buffer, dma_element_type** __dma_dest, dma_element_type** __dma_source)
__device__ void __loadParametersToShm()
{
	int32 local_tid = threadIdx.x - CUDA_COMPUTE_THREADS_PER_BLOCK;
	int32 shuffled_index = blockIdx.x * CUDA_COMPUTE_THREADS_PER_BLOCK + local_tid;

	__dma_source[local_tid] = (dma_element_type*)&gd_current_invocation_buffer_ptr[shuffled_index];
	__dma_dest[local_tid] = (dma_element_type*)&__parameter_shm[local_tid * CUDA_INVOCATION_PARAMETER_SIZE];
	__dma_size[local_tid] = CUDA_INVOCATION_PARAMETER_SIZE;

	__barrierArrive(CUDA_DMA_EMPTY_BARRIER, CUDA_DISPATCHER_THREADS_PER_BLOCK);
	__barrierSync(CUDA_DMA_FULL_BARRIER, CUDA_DISPATCHER_THREADS_PER_BLOCK);
}

__device__ void __loadParametersToShm(char* shm)
{
//	if(threadIdx.x == 0)
//	{
//		char* parameters = (char*)gd_current_invocation_buffer_ptr;
//		param_element_type* typed_shm = (param_element_type*)__parameter_shm;
//		param_element_type* typed_parameters = (param_element_type*)parameters;
//		printf("%d: load param from %p\n", threadIdx.x, parameters);
//		for(int i=0;i<CUDA_COMPUTE_THREADS_PER_BLOCK*CUDA_INVOCATION_PARAMETER_SIZE/CUDA_INVOCATION_PARAMETER_LOAD_SIZE;++i)
//		{
//			typed_shm[i] = typed_parameters[i];
//		}
//	}

	char* parameters = (char*)gd_current_invocation_buffer_ptr;
	int32* shuffle_indices = gd_current_invocation_shuffle_indices_ptr;
	int32 total_invocation = gd_current_invocation_buffer_size;

	//int tid = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("blockIdx.x = %d, blockDim.x = %d, tid = %d\n", blockIdx.x, blockDim.x, tid);
	int32 warp_id = threadIdx.x / CUDA_DISPATCHER_WARP_SIZE;
	int32 wid = threadIdx.x % CUDA_DISPATCHER_WARP_SIZE;
	int32 base_id = blockIdx.x * blockDim.x;

	param_element_type* typed_shm = (param_element_type*)shm;
	param_element_type* typed_parameters = (param_element_type*)parameters;

#if __TOTAL_LOADER_WARP_COUNT == CUDA_DISPATCHER_WARP_SIZE && false
	int32 shuffled_index = -1;
	int32 current_row = base_id;

	#pragma unroll 32
	for(int j=0;j<__TOTAL_WORKING_ITEMS_PER_WARP;++j)
	{
		if(current_row >= total_invocation)
			break;

		shuffled_index = shuffle_indices[current_row];

		// load from global memory into shared memory
		int32 to_index = j * __INVOCATION_PARAMETER_ELEMENTS + threadIdx.x;
		int32 from_index = shuffled_index * __INVOCATION_PARAMETER_ELEMENTS + threadIdx.x;
		//printf("tid=%d, shuffled_index=%d, to_idx=%d, from_idx=%d\n", tid, shuffled_index, to_index, from_index);
		typed_shm[to_index] = typed_parameters[from_index];

		// load next item
		++current_row;
	}
#else
	int32 shuffled_index = -1;
	int32 last_row = -1;
	int32 working_index = warp_id;
	int32 current_row = base_id + working_index / __TOTAL_WARPS_PER_INVOCATION;

	#pragma unroll 32
	for(int j=0;j<__TOTAL_WORKING_ITEMS_PER_WARP;++j)
	{
		if(current_row >= total_invocation)
			break;

		if(current_row != last_row)
		{
			shuffled_index = shuffle_indices[current_row];
		}

		// load from global memory into shared memory
		int32 local_index = working_index % __TOTAL_WARPS_PER_INVOCATION;
		int32 to_index = working_index * CUDA_DISPATCHER_WARP_SIZE + wid;
		int32 from_index = shuffled_index * __INVOCATION_PARAMETER_ELEMENTS + local_index * CUDA_DISPATCHER_WARP_SIZE + wid;
		//printf("tid=%d, wid=%d, warp_id=%d, working_idx=%d, current_row=%d, shuffled_idx=%d, local_index=%d, to_idx=%d, from_idx=%d\n", base_id + threadIdx.x, wid, warp_id, working_index, current_row, shuffled_index, local_index, to_index, from_index);
		typed_shm[to_index] = typed_parameters[from_index];

		// load next item
		working_index += __TOTAL_WARPS_PER_BLOCK;
		last_row = current_row;
		current_row = base_id + working_index / __TOTAL_WARPS_PER_INVOCATION;
	}
#endif
}

__device__ bool __isDMAWarp()
{
	return threadIdx.x < CUDA_DMA_THREADS_PER_BLOCK;
}

__device__ void __runDMAWarp()
{
	while(true)
	{
		__barrierSync(CUDA_DMA_EMPTY_BARRIER, CUDA_DISPATCHER_THREADS_PER_BLOCK);

		bool has_dma = false;
		for(int i=0;i<CUDA_COMPUTE_THREADS_PER_BLOCK;++i)
		{
			int32 size;
			if((size = __dma_size[i]) > 0)
			{
//				if(threadIdx.x == 0)
//					printf("%d (%d): transfer from %p to %p (size = %d, i = %d)\n", threadIdx.x, blockIdx.x, __dma_source[i], __dma_dest[i], size, i);

				has_dma = true;
				__dma_dest[i][threadIdx.x] = __dma_source[i][threadIdx.x];
				if(threadIdx.x == 0)
					__dma_size[i] = 0;
			}
		}

		if(!has_dma) break;

		__barrierArrive(CUDA_DMA_FULL_BARRIER, CUDA_DISPATCHER_THREADS_PER_BLOCK);
	}
}

__device__ void __initDMA()
{
	int32 local_tid = threadIdx.x - CUDA_DMA_THREADS_PER_BLOCK;
	__dma_size[local_tid] = 0;
}

__device__ void __startDMA(int32 target_id, int32 invocation_id)
{
	using namespace zillians::framework::service;

//	printf("start DMA, target id = %d, invocation id = %d\n", target_id, invocation_id);
//	return;

	int32 local_tid = threadIdx.x - CUDA_DMA_THREADS_PER_BLOCK;
	int bucket_id = blockIdx.x % CUDA_INVOCATION_NEXT_BUCKET_SIZE;

	__dma_source[local_tid] = &__dma_buffer[local_tid*CUDA_INVOCATION_DMA_BUFFER_SIZE_PER_THREAD/CUDA_INVOCATION_DMA_TRANSFER_SIZE];
	__dma_dest[local_tid] = (dma_element_type*)&((Invocation*)gd_next_invocation_buffer_ptr[target_id * CUDA_INVOCATION_NEXT_BUCKET_SIZE + bucket_id])[invocation_id];
	__dma_size[local_tid] = CUDA_INVOCATION_DMA_BUFFER_SIZE_PER_THREAD;

//	__threadfence_block();

	__barrierArrive(CUDA_DMA_EMPTY_BARRIER, CUDA_DISPATCHER_THREADS_PER_BLOCK);
	__barrierSync(CUDA_DMA_FULL_BARRIER, CUDA_DISPATCHER_THREADS_PER_BLOCK);

//	__dma_size[local_tid] = 0;
}

__device__ void __completeDMA()
{
	__dma_size[threadIdx.x - CUDA_DMA_THREADS_PER_BLOCK] = 0;

//	__threadfence_block();

	__barrierArrive(CUDA_DMA_EMPTY_BARRIER, CUDA_DISPATCHER_THREADS_PER_BLOCK);
//	__barrierSync(CUDA_DMA_FULL_BARRIER, CUDA_DISPATCHER_THREADS_PER_BLOCK);
}

}

__device__ void daemonize()
{
    internal::__control_flags->daemonized = true;
}

__device__ void exit(int32 exit_code)
{
    internal::__control_flags->explicit_exited = true;
    internal::__control_flags->explicit_exit_code = exit_code;
}

__device__ int32 __getCurrentInvocationId()
{
	//
	// 0, 1, ..., CUDA_DMA_THREADS_PER_BLOCK, CUDA_DMA_THREADS_PER_BLOCK+1, ..., CUDA_DMA_THREADS_PER_BLOCK+CUDA_COMPUTE_THREADS_PER_BLOCK
	// <----------DMA threads---------------> <---------------------------------compute threads------------------------------------------>
	//
	int32 local_id = threadIdx.x - CUDA_DMA_THREADS_PER_BLOCK;
	int32 invoke_id = blockIdx.x * CUDA_COMPUTE_THREADS_PER_BLOCK + local_id;
	if(invoke_id >= internal::gd_current_invocation_buffer_size)
		return -1;
	else
		return local_id;
}

//__device__ int64 __getCurrentFunctionId(char* shm, int32 invocation_id)
__device__ int64 __getCurrentFunctionId(int32 invocation_id)
{

	return *(int64*)(((char*)internal::__parameter_shm) + invocation_id * CUDA_INVOCATION_PARAMETER_SIZE);
}

//__device__ int64 __getCurrentSessionId(char* shm, int32 invocation_id)
__device__ int64 __getCurrentSessionId(int32 invocation_id)
{
	return *(int64*)(((char*)internal::__parameter_shm) + invocation_id * CUDA_INVOCATION_PARAMETER_SIZE + sizeof(int64));
}

//__device__ char* __getCurrentParameterPtr(char* shm, int32 invocation_id)
__device__ char* __getCurrentParameterPtr(int32 invocation_id)
{
	return (char*)(((char*)internal::__parameter_shm) + invocation_id * CUDA_INVOCATION_PARAMETER_SIZE + sizeof(int64) + sizeof(int64) + sizeof(int64));
}

//__device__ void __setLocalDomainObject(Object* domain)
//{
////    gRuntimeServiceBuffer->setLocalDomainObject(domain);
//}
//
//__device__ void __addDomainObject(int64 session_id, Object* domain)
//{
////    gRuntimeServiceBuffer->addDomainObject(session_id, domain);
//}
//
//__device__ Domain* __findDomainObject(int64 session_id)
//{
////    return (Domain*)gRuntimeServiceBuffer->findDomainObject(session_id);
//}
//
//__device__ void __getDomainCallbacks(int32 event, container::Vector< Lambda1<void, Domain> >* callbacks )
//{
////    gRuntimeServiceBuffer->getDomainCallbacks(event, callbacks);
//}

__device__ void  __setImplicitExitCode(int32 exit_code)
{
    internal::__control_flags->implicit_exit_code = exit_code;
    internal::__control_flags->implicit_exited = true;
}

__device__ int32 __getNextInvocationId(int32 target_id)
{
	int bucket_id = blockIdx.x % CUDA_INVOCATION_NEXT_BUCKET_SIZE;
	int index = target_id * CUDA_INVOCATION_NEXT_BUCKET_SIZE + bucket_id;
	int32 next_invocation_id = atomicAdd(&internal::gd_next_invocation_buffer_index[index], 1);
	if(next_invocation_id >= internal::gd_next_invocation_buffer_size[index])
	{
		// TODO assert here
		return -1;
	}
	return next_invocation_id;
}

__device__ void __setNextFunctionId(int32 target_id, int32 invocation_id, int64 function_id)
{
	using namespace zillians::framework::service;

	int32 local_tid = threadIdx.x - CUDA_DMA_THREADS_PER_BLOCK;
	char* dma_buffer_ptr = (char*)internal::__dma_buffer;
	dma_buffer_ptr += local_tid * CUDA_INVOCATION_DMA_BUFFER_SIZE_PER_THREAD;
	Invocation* invocation = (Invocation*)dma_buffer_ptr;
//	printf("%d (%d): set next function id = %ld (%p, %p, %d)\n", threadIdx.x, blockIdx.x, function_id, (char*)internal::__dma_buffer, invocation, local_tid * CUDA_INVOCATION_DMA_BUFFER_SIZE_PER_THREAD);
	invocation->function_id = function_id;

}

__device__ char* __getNextParameterPtr(int32 target_id, int32 invocation_id)
{
	using namespace zillians::framework::service;

	int32 local_tid = threadIdx.x - CUDA_DMA_THREADS_PER_BLOCK;
	return (char*)(&((Invocation*)(&internal::__dma_buffer[local_tid * CUDA_INVOCATION_DMA_BUFFER_SIZE_PER_THREAD / CUDA_INVOCATION_DMA_TRANSFER_SIZE]))->parameters[0]);
}

__device__ void __invokeFunction(Object* domain, int64 function_id)
{
	//printf("%d: invoke function %p, function id = %ld\n", threadIdx.x, domain, function_id);
	//if(domain)
	{
		//int64 session_id = domain->domain()->session_id;
		//int32 target_id = session_id >> 32;
		int32 target_id = 0;

		int32 invocation_id = __getNextInvocationId(target_id);
//		printf("%d: next invocation id = %d\n", threadIdx.x, invocation_id);

		__setNextFunctionId(target_id, invocation_id, function_id);

//		printf("%d: before start DMA\n", threadIdx.x);
		internal::__startDMA(target_id, invocation_id);
	}
}

} }

